#include "hip/hip_runtime.h"
#include "siluPlugin.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void SiLUKernel(size_t input_size, const T* input, T* output) {
    size_t index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= input_size) {
        return;
    }

    output[index] = input[index] / (1 + expf(-input[index]));
}

template <typename T>
pluginStatus_t inferenceSiLU(int size, const T* input, T* output, hipStream_t stream) {
    const int blockSize = 512;
    const int gridSize = (size + blockSize - 1) / blockSize;
    SiLUKernel<T><<<gridSize, blockSize, 0, stream>>>(size, input, output);
    return STATUS_SUCCESS;
}

int SiLUPlugin::enqueue(int batchSize, const void* const* inputs, void** outputs, void* workspace, hipStream_t stream)
{
    switch (mDataType)
    {
    case DataType::kFLOAT:
        return inferenceSiLU(batchSize * mBatchDim, (float*)inputs[0], (float*)outputs[0], stream);
    case DataType::kINT32:
        return inferenceSiLU(batchSize * mBatchDim, (int32_t*)inputs[0], (int32_t*)outputs[0], stream);
    case DataType::kINT8:
        return inferenceSiLU(batchSize * mBatchDim, (int8_t*)inputs[0], (int8_t*)outputs[0], stream);
    }
    return 1;
}
